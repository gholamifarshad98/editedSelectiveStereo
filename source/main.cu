#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////
/// At first you must set stack commit and stack resservd to 78125000
////////////////////////////////////////////////////////////////////
#pragma once
#include <hip/hip_cooperative_groups.h>
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include<iostream>
#include<math.h>
#include<vector>
#include<memory>
#include <chrono> 
#include<string> 
#include<math.h>
#include<fstream>
#include<hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <time.h>
#include<stdio.h>
#include<iostream> 
#include<algorithm>



using namespace std;
using namespace cv;


int numOfColumnsResized;
int numOfRowsResized = 0;
int kernelSize = 16;
int maxDisparity = 30;
int selectedDisparity =5;


__global__ void IDAS_Stereo_selective( int MxDisparity, int nC , int nSelected, uchar* leftIm, uchar* rightIm, int* resultIm)
{
	//thread_group my_block = this_thread_block();
	//thread_block block = this_thread_block();
	__shared__   int costs;
	__shared__   int dif[16 * 16];
	int kSize = 16;
	

	int rightPixelIndexU;
	int rightPixelIndexV;

	int leftPixelIndexU;
	int leftPixelIndexV;

	int leftPixelIndex;
	int rightPixelIndex;

	rightPixelIndexU = blockIdx.x + threadIdx.x + MxDisparity + 1;
	rightPixelIndexV = blockIdx.y + threadIdx.y;
	leftPixelIndexU = rightPixelIndexU + (blockIdx.z - 1) + nSelected;
	leftPixelIndexV = rightPixelIndexV;
	leftPixelIndex = leftPixelIndexV * nC + leftPixelIndexU;
	rightPixelIndex = rightPixelIndexV * nC + rightPixelIndexU;
	dif[threadIdx.x + threadIdx.y * kSize] = abs(leftIm[leftPixelIndex] - rightIm[rightPixelIndex]);
	__syncthreads();

	
	if (threadIdx.x == 0 & threadIdx.y == 0) {
		costs = 0;
		for (int i = 0; i < blockDim.y;i++) {
			for(int j=0;j< blockDim.x;j++)
			costs = costs + dif[i + j* kSize];
		}
		//printf("%d \n", costs);
	}

	__syncthreads();

			
	resultIm[((blockIdx.y + int(kSize / 2))* nC + (blockIdx.x + MxDisparity+ int(kSize / 2)+2))*3+ blockIdx.z] = costs;
	
}
void ReadBothImages(shared_ptr<Mat> leftImage, shared_ptr<Mat> rightImage,int* numOfRows,int* numOfColumns) {

	try {
		//cout << "this is test" << endl;
		*rightImage = cv::imread("2.png", IMREAD_GRAYSCALE);   // Read the right image
																  //rightImage->convertTo(*rightImage, CV_64F);
		*leftImage = cv::imread("1.png", IMREAD_GRAYSCALE);   // Read the left image
		*numOfColumns = ((int)leftImage->cols / 32) * 32;
		*numOfRows = ((int)leftImage->rows / 32) * 32;													 //leftImage->convertTo(*leftImage, CV_64F);
	}
	catch (char* error) {
		cout << "can not load the " << error << " iamge" << endl;
	}

	//cv::resize(*rightImage, *rightImage, cv::Size(), 0.50, 0.50);
	//cv::resize(*leftImage, *leftImage, cv::Size(), 0.50, 0.50);

	//imshow("test", *rightImage);


	//waitKey(5000);
}
int CalcCost(shared_ptr<Mat> leftImage_, shared_ptr<Mat> rightImage_, int row, int column, int kernelSize, int disparity, int NCols) {
	int cost = 0;
	
	for (int u = -int(kernelSize / 2); u <= int(kernelSize / 2); u++) {
		for (int v = -int(kernelSize / 2); v <= int(kernelSize / 2); v++) {
			// for error handeling.
			if (column + u + disparity >= NCols) {
				cout << "*****************************************************" << endl;
			}
			cost = cost + int(pow((leftImage_->at<uchar>(row + v, column + u) - (rightImage_->at<uchar>(row + v, column + u + disparity))), 2));
		}
	}
	return cost;
}
void  SSDstereo(shared_ptr<Mat> leftImage_, shared_ptr<Mat> rightImage_, shared_ptr<Mat> result_temp_, int kernelSize, int maxDisparity, int NRow, int NCols) {
	int tempCost = 0;
	int tempDisparity = 0;

	for (int u = (kernelSize / 2) + 1; u <(NCols - maxDisparity - kernelSize / 2) - 1; u++) {
		for (int v = (kernelSize / 2) + 1; v <NRow - (kernelSize / 2); v++) {
			double cost = 10000000;
			tempCost = 0;
			tempDisparity = 0;
			for (int i = 0; i < maxDisparity; i++) {
				tempCost = CalcCost(leftImage_, rightImage_, v, u, kernelSize, i, NCols);
				if (tempCost < cost) {
					cost = tempCost;
					tempDisparity = i;
				}
			}
			tempDisparity = tempDisparity * 255 / maxDisparity;
			result_temp_->at<uchar>(v, u) = tempDisparity;
			//std::cout << " tempDisparity for ("<< u<<","<<v<<") is "  << tempDisparity << std::endl;
		}
	}
	//std::cout << "debug" << std::endl;
	//cv::imshow("stereoOutput", *result_temp);
	//cv::waitKey(100);
}

int main(void)
{
	//=======================================================================================================================================
	//Memeory Alocation
	//=======================================================================================================================================
	chrono::high_resolution_clock::time_point startTimeReadImage;
	chrono::high_resolution_clock::time_point stopTimeReadImage;
	std::chrono::duration<double, std::milli> durationReadImage;

	chrono::high_resolution_clock::time_point startConvertTo1D;
	chrono::high_resolution_clock::time_point stopConvertTo1D;
	std::chrono::duration<double, std::milli> durationConvertTo1D;

	chrono::high_resolution_clock::time_point startCudaMemcpyInput;
	chrono::high_resolution_clock::time_point stopCudaMemcpyInput;
	std::chrono::duration<double, std::milli> durationCudaMemcpyInput;

	chrono::high_resolution_clock::time_point startCudaCalc;
	chrono::high_resolution_clock::time_point stopCudaCalc;
	std::chrono::duration<double, std::milli> durationCudaCalc;

	chrono::high_resolution_clock::time_point startCudaMemcpyResult;
	chrono::high_resolution_clock::time_point stopCudaMemcpyResult;
	std::chrono::duration<double, std::milli> durationCudaMemcpyResult;

	chrono::high_resolution_clock::time_point startInferenceResult;
	chrono::high_resolution_clock::time_point stopInferenceResult;
	std::chrono::duration<double, std::milli> durationInferenceResult;
	std::chrono::duration<double, std::milli> totalDuraation;



	shared_ptr<Mat> rightImage = make_shared<Mat>();
	shared_ptr<Mat> leftImage = make_shared<Mat>();

	shared_ptr<Mat> rightImageResized = make_shared<Mat>();
	shared_ptr<Mat> leftImageResized = make_shared<Mat>();

	shared_ptr<Mat>  stereoResut = make_shared<Mat>();
	shared_ptr<Mat>  stereoResutResized = make_shared<Mat>();

	int numOfRows;
	int numOfColumns;
	
	//Object for repoting results in a text file.
	ofstream repotringResult;


	//Varaible for convert 2D images to 1D array of images.
	uchar** imArray2DL;
	uchar** imArray2DR;
	int** imArrary2DR_result;
	uchar* imArrary1DL;
	uchar* imArrary1DR;
	int* imArrary1DR_result;
	
	//Varaible for inference the results;
	int firstCost;
	int secondCost;
	int thirdCost;

	//Pointers for Memeory Alocation on GPU.
	uchar* imArray1DL_d;
	uchar* imArray1DR_d;
	int* imArray1DResult_d;




	//=======================================================================================================================================
	//Read Image
	//=======================================================================================================================================
	startTimeReadImage  = chrono::high_resolution_clock::now();
	ReadBothImages(leftImage, rightImage,&numOfRows,&numOfColumns);
	stopTimeReadImage = chrono::high_resolution_clock::now();



	
	

	//=======================================================================================================================================
	//Dynamic Memeory Alocation 
	//=======================================================================================================================================
	stereoResut = make_shared<Mat>(numOfRows, numOfColumns, CV_8UC1);
	imArray2DL= new uchar* [numOfRows];
	imArray2DR = new uchar*[numOfRows];
	imArrary2DR_result= new int*[numOfRows];
	for (int i = 0; i < numOfRows; i++) {
		imArray2DL[i] = new uchar[numOfColumns];
		imArray2DR[i] = new uchar[numOfColumns];
		imArrary2DR_result[i] = new int[numOfColumns*3];
	}
	imArrary1DL = new uchar[numOfColumns * numOfRows];
	imArrary1DR = new uchar[numOfColumns * numOfRows];
	imArrary1DR_result = new int[numOfColumns * numOfRows * 3];
	
	
	hipMalloc((void**)&imArray1DL_d, numOfColumns * numOfRows * sizeof(uchar));
	hipMalloc((void**)&imArray1DR_d, numOfColumns * numOfRows * sizeof(uchar));
	hipMalloc((void**)&imArray1DResult_d, numOfColumns * numOfRows * 3 * sizeof(int));

	// Set grid and bolck size.
	dim3 blocks3D(16, 16, 1);
	dim3 grid2D(numOfColumns - 2 * (maxDisparity + 1) - (kernelSize - 1), numOfRows - kernelSize - 1, 3);





	//=======================================================================================================================================
	//Convert 2D image To 1D  array.
	//=======================================================================================================================================
	startConvertTo1D = chrono::high_resolution_clock::now();
	for (int j = 0; j < numOfRows; j++) {
		for (int i = 0; i < numOfColumns; i++) {
			imArray2DL[j][i] = leftImage->at<uchar>(j, i);
			imArray2DR[j][i] = rightImage->at<uchar>(j, i);
		}
	}
	
	for (int i = 0; i < numOfColumns*numOfRows; i++) {
		imArrary1DL[i] = imArray2DL[int(i / numOfColumns)][i%numOfColumns];
		imArrary1DR[i] = imArray2DR[int(i / numOfColumns)][i%numOfColumns];
		for(int k=0;k<3;k++){
			imArrary1DR_result[i + i*k] = 0;
		}
	}
	stopConvertTo1D = chrono::high_resolution_clock::now();



	

	//=======================================================================================================================================
	//Copy 1D images to GPU.
	//=======================================================================================================================================
	startCudaMemcpyInput = chrono::high_resolution_clock::now();
	hipMemcpy(imArray1DL_d, imArrary1DL, numOfColumns*numOfRows * sizeof(uchar), hipMemcpyHostToDevice);
	hipMemcpy(imArray1DR_d, imArrary1DR, numOfColumns*numOfRows * sizeof(uchar), hipMemcpyHostToDevice);
	stopCudaMemcpyInput = chrono::high_resolution_clock::now();
	



	//=======================================================================================================================================
	//Call kernel to run on GPU.
	//=======================================================================================================================================
	startCudaCalc = chrono::high_resolution_clock::now();
	IDAS_Stereo_selective <<<grid2D, blocks3D >>>(maxDisparity, numOfColumns, selectedDisparity, imArray1DL_d, imArray1DR_d, imArray1DResult_d);
	stopCudaCalc = chrono::high_resolution_clock::now();
	
	


	//=======================================================================================================================================
	//Copy 1D result from GPU.
	//=======================================================================================================================================
	startCudaMemcpyResult = chrono::high_resolution_clock::now();
	hipMemcpy(imArrary1DR_result, imArray1DResult_d, numOfColumns*numOfRows *3* sizeof(int), hipMemcpyDeviceToHost);
	stopCudaMemcpyResult = chrono::high_resolution_clock::now();





	
	//=======================================================================================================================================
	//Inference Results.
	//=======================================================================================================================================
	startInferenceResult = chrono::high_resolution_clock::now();
	for (int j = 0; j < numOfRows; j++) {
		for (int i = 0; i < numOfColumns; i++) {
			firstCost = imArrary1DR_result[(j * numOfColumns + i)*3];
			secondCost= imArrary1DR_result[(j * numOfColumns + i)*3+1];
			thirdCost= imArrary1DR_result[(j * numOfColumns + i )* 3+2];
			if(secondCost<firstCost& secondCost<thirdCost)
				leftImage->at<uchar>(j, i)=(uchar)255 ;
		}
	}
	stopInferenceResult = chrono::high_resolution_clock::now();

	



	//=======================================================================================================================================
	//Memeory De-alocation.
	//=======================================================================================================================================
	
	hipFree(imArray1DL_d);
	hipFree(imArray1DR_d);
	hipFree(imArray1DResult_d);
	delete imArray2DL;
	delete imArray2DR;
	delete imArrary2DR_result;
	delete imArrary1DL;
	delete imArrary1DR;
	delete imArrary1DR_result;

	





	//=======================================================================================================================================
	//Reporting the results.
	//=======================================================================================================================================
	durationReadImage = stopTimeReadImage - startTimeReadImage;
	durationConvertTo1D= stopConvertTo1D- startConvertTo1D;
	durationCudaMemcpyInput= stopCudaMemcpyInput- startCudaMemcpyInput;
	durationCudaCalc = stopCudaCalc - startCudaCalc;
	durationCudaMemcpyResult = stopCudaMemcpyResult - startCudaMemcpyResult;
	durationInferenceResult = stopInferenceResult - startInferenceResult;
	totalDuraation = durationReadImage + durationConvertTo1D + durationCudaMemcpyInput +
		durationCudaCalc + durationCudaMemcpyResult+ durationInferenceResult;

	string durationReadImage_s = to_string(durationReadImage.count());
	string durationConvertTo1D_s = to_string(durationConvertTo1D.count());
	string durationCudaMemcpyInput_s = to_string(durationCudaMemcpyInput.count());
	string durationCudaCalc_s = to_string(durationCudaCalc.count());
	string durationCudaMemcpyResult_s = to_string(durationCudaMemcpyResult.count());
	string durationInferenceResult_s = to_string(durationInferenceResult.count());
	string totalDuraation_s = to_string(totalDuraation.count());

	repotringResult.open("results.txt");
	repotringResult << "durationReadImage = " << durationReadImage_s << endl;
	repotringResult << "durationConvertTo1D = " << durationConvertTo1D_s << endl;
	repotringResult << "durationCudaMemcpyInput = " << durationCudaMemcpyInput_s << endl;
	repotringResult << "durationCudaCalc = " << durationCudaCalc_s << endl;
	repotringResult << "durationCudaMemcpyResult = " << durationCudaMemcpyResult_s << endl;
	repotringResult << "durationInferenceResult = " << durationInferenceResult_s << endl;
	repotringResult << "totalDuraation = " << totalDuraation_s << endl;
	repotringResult.close();




	imshow(" Left after calaculation !!!", *leftImage);
	imwrite("result.png", *leftImage);
	imshow("Right image !!!   .....", *rightImage);
	waitKey(1000);
	printf("\n \n \n  \t \t \t :)  ");
	char str[80];
	scanf("%79s", str);
}




//command for syncronization of thread.
//hipDeviceSynchronize();

//copy data to 2d reasult image.
	/*for (int i = 0; i < numOfColumns*numOfRows; i++) {
		imArrary2DR_result[int(i / numOfColumns)][i%numOfColumns] = imArrary1DR_result[i];
	}*/